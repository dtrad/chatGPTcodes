// nvcc -arch=sm_86 maxmult2.cu -o maxmult2
#include <hip/hip_runtime.h>
#include <iostream>
#include <cassert>

#define CUDA_CHECK(call) \
    { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ << " code=" << err << " \"" << hipGetErrorString(err) << "\"" << std::endl; \
            exit(1); \
        } \
    }

__global__ void matrixMultiplyKernel(float *A, float *B, float *C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < N && col < N) {
        float value = 0.0;
        for (int k = 0; k < N; ++k) {
            value += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = value;
    }
}

void matrixMultiply(float *A, float *B, float *C, int N) {
    size_t size = N * N * sizeof(float);

    float *d_A, *d_B, *d_C;

    CUDA_CHECK(hipMalloc(&d_A, size));
    CUDA_CHECK(hipMalloc(&d_B, size));
    CUDA_CHECK(hipMalloc(&d_C, size));

    CUDA_CHECK(hipMemcpy(d_A, A, size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, B, size, hipMemcpyHostToDevice));

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((N + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (N + threadsPerBlock.y - 1) / threadsPerBlock.y);

    matrixMultiplyKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    CUDA_CHECK(hipGetLastError());

    CUDA_CHECK(hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost));

    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_C));
}

void verifyResult(float *A, float *B, float *C, int N) {
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            float value = 0;
            for (int k = 0; k < N; ++k) {
                value += A[i * N + k] * B[k * N + j];
            }
            assert(fabs(C[i * N + j] - value) < 1e-2);
        }
    }
}

int main() {
    int N = 1024;
    size_t size = N * N * sizeof(float);

    float *A = (float *)malloc(size);
    float *B = (float *)malloc(size);
    float *C = (float *)malloc(size);

    // Initialize matrices A and B
    for (int i = 0; i < N * N; ++i) {
        A[i] = static_cast<float>(rand()) / RAND_MAX;
        B[i] = static_cast<float>(rand()) / RAND_MAX;
    }

    matrixMultiply(A, B, C, N);

    // Verify the result
    verifyResult(A, B, C, N);

    std::cout << "Matrix multiplication successful and verified." << std::endl;

    free(A);
    free(B);
    free(C);

    return 0;
}
