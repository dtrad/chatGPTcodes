#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

#define CUDA_CHECK(call)                                                                        \
    do {                                                                                         \
        hipError_t err__ = (call);                                                              \
        if (err__ != hipSuccess) {                                                              \
            std::cerr << "CUDA error: " << hipGetErrorString(err__) << " at " << __FILE__      \
                      << ":" << __LINE__ << std::endl;                                          \
            std::exit(EXIT_FAILURE);                                                              \
        }                                                                                        \
    } while (0)

__global__ void scaleVector(const float *input, float *output, float factor, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        output[idx] = input[idx] * factor;
    }
}

int main() {
    const int elementCount = 16;
    const size_t bytes = elementCount * sizeof(float);
    const float scale = 1.5f;

    std::vector<float> hostInput(elementCount);
    std::vector<float> hostOutput(elementCount, 0.0f);

    for (int i = 0; i < elementCount; ++i) {
        hostInput[i] = static_cast<float>(i);
    }

    float *deviceInput = nullptr;
    float *deviceOutput = nullptr;

    CUDA_CHECK(hipMalloc(&deviceInput, bytes));
    CUDA_CHECK(hipMalloc(&deviceOutput, bytes));

    CUDA_CHECK(hipMemcpy(deviceInput, hostInput.data(), bytes, hipMemcpyHostToDevice));

    const int threadsPerBlock = 128;
    const int blocksPerGrid = (elementCount + threadsPerBlock - 1) / threadsPerBlock;

    scaleVector<<<blocksPerGrid, threadsPerBlock>>>(deviceInput, deviceOutput, scale, elementCount);
    CUDA_CHECK(hipGetLastError());

    CUDA_CHECK(hipMemcpy(hostOutput.data(), deviceOutput, bytes, hipMemcpyDeviceToHost));

    std::cout << "Scaled vector:" << std::endl;
    for (int i = 0; i < elementCount; ++i) {
        std::cout << "  " << hostInput[i] << " -> " << hostOutput[i] << std::endl;
    }

    CUDA_CHECK(hipFree(deviceInput));
    CUDA_CHECK(hipFree(deviceOutput));

    return 0;
}
