#include <hip/hip_runtime.h>
#include <iostream>

__global__ void matrixMultiplyKernel(float *A, float *B, float *C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < N && col < N) {
        float value = 0.0;
        for (int k = 0; k < N; ++k) {
            value += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = value;
    }
}

void matrixMultiply(float *A, float *B, float *C, int N) {
    size_t size = N * N * sizeof(float);

    float *d_A, *d_B, *d_C;

    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((N + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (N + threadsPerBlock.y - 1) / threadsPerBlock.y);

    matrixMultiplyKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

int main() {
    int N = 1024;
    size_t size = N * N * sizeof(float);

    float *A = (float *)malloc(size);
    float *B = (float *)malloc(size);
    float *C = (float *)malloc(size);

    // Initialize matrices A and B
    for (int i = 0; i < N * N; ++i) {
        A[i] = static_cast<float>(rand()) / RAND_MAX;
        B[i] = static_cast<float>(rand()) / RAND_MAX;
    }

    matrixMultiply(A, B, C, N);

    // Print some of the result
    std::cout << "C[0] = " << C[0] << std::endl;
    std::cout << "C[N*N-1] = " << C[N*N-1] << std::endl;

    free(A);
    free(B);
    free(C);

    return 0;
}
