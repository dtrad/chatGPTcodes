// write below the command line necessary to compile this code
// nvcc -o testcodex testcodex.cu -arch=sm_86
// create a cuda example with codex
#include <iostream>
#include <hip/hip_runtime.h>

__global__ void addKernel(int *c, const int *a, const int *b, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        c[i] = a[i] + b[i];
    }
}

int main() {
    const int arraySize = 5;
    int a[arraySize] = {1, 2, 3, 4, 5};
    int b[arraySize] = {10, 20, 30, 40, 50};
    int c[arraySize] = {0};

    int *dev_a, *dev_b, *dev_c;
    hipMalloc((void**)&dev_a, arraySize * sizeof(int));
    hipMalloc((void**)&dev_b, arraySize * sizeof(int));
    hipMalloc((void**)&dev_c, arraySize * sizeof(int));

    hipMemcpy(dev_a, a, arraySize * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, arraySize * sizeof(int), hipMemcpyHostToDevice);

    // some elements are incorrect, fix
    
    // use several blocks
    int blockSize = 2;
    int numBlocks = (arraySize + blockSize - 1) / blockSize;
    addKernel<<<numBlocks, blockSize>>>(dev_c, dev_a, dev_b, arraySize);

    hipMemcpy(c, dev_c, arraySize * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < arraySize; i++) {
        std::cout << c[i] << " ";
    }
    std::cout << std::endl;

    // add comparison
    for (int i = 0; i < arraySize; i++) {
        if (c[i] == a[i] + b[i]) {
            std::cout << "Element " << i << " is correct." << std::endl;
        } else {
            std::cout << "Element " << i << " is incorrect." << std::endl;
        }
    }
    

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}
