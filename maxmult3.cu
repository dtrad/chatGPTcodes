// compile with nvcc -arch=sm_86 maxmult3.cu -o maxmult3
#include <hip/hip_runtime.h>
#include <iostream>
#include <cassert>
#include <cmath>

#define CUDA_CHECK(call) \
    { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ << " code=" << err << " \"" << hipGetErrorString(err) << "\"" << std::endl; \
            exit(1); \
        } \
    }

__global__ void matrixMultiplyKernel(float *A, float *B, float *C, int M, int N, int P) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < M && col < P) {
        float value = 0.0;
        for (int k = 0; k < N; ++k) {
            value += A[row * N + k] * B[k * P + col];
        }
        C[row * P + col] = value;
    }
}

void matrixMultiply(float *A, float *B, float *C, int M, int N, int P) {
    size_t sizeA = M * N * sizeof(float);
    size_t sizeB = N * P * sizeof(float);
    size_t sizeC = M * P * sizeof(float);

    float *d_A, *d_B, *d_C;

    CUDA_CHECK(hipMalloc(&d_A, sizeA));
    CUDA_CHECK(hipMalloc(&d_B, sizeB));
    CUDA_CHECK(hipMalloc(&d_C, sizeC));

    CUDA_CHECK(hipMemcpy(d_A, A, sizeA, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, B, sizeB, hipMemcpyHostToDevice));

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((P + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (M + threadsPerBlock.y - 1) / threadsPerBlock.y);

    matrixMultiplyKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, M, N, P);
    CUDA_CHECK(hipGetLastError());

    CUDA_CHECK(hipMemcpy(C, d_C, sizeC, hipMemcpyDeviceToHost));

    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_C));
}

void verifyResult(float *A, float *B, float *C, int M, int N, int P) {
    for (int i = 0; i < M; ++i) {
        for (int j = 0; j < P; ++j) {
            float value = 0;
            for (int k = 0; k < N; ++k) {
                value += A[i * N + k] * B[k * P + j];
            }
            assert(fabs(C[i * P + j] - value) < 1e-2);
        }
    }
}

int main() {
    // Test 1: Square matrices
    int M1 = 1024, N1 = 1024, P1 = 1024;
    size_t size1 = M1 * N1 * sizeof(float);
    float *A1 = (float *)malloc(size1);
    float *B1 = (float *)malloc(size1);
    float *C1 = (float *)malloc(size1);
    for (int i = 0; i < M1 * N1; ++i) {
        A1[i] = static_cast<float>(rand()) / RAND_MAX;
        B1[i] = static_cast<float>(rand()) / RAND_MAX;
    }
    matrixMultiply(A1, B1, C1, M1, N1, P1);
    verifyResult(A1, B1, C1, M1, N1, P1);
    free(A1);
    free(B1);
    free(C1);

    // Test 2: Non-square matrices
    int M2 = 512, N2 = 1024, P2 = 768;
    size_t sizeA2 = M2 * N2 * sizeof(float);
    size_t sizeB2 = N2 * P2 * sizeof(float);
    size_t sizeC2 = M2 * P2 * sizeof(float);
    float *A2 = (float *)malloc(sizeA2);
    float *B2 = (float *)malloc(sizeB2);
    float *C2 = (float *)malloc(sizeC2);
    for (int i = 0; i < M2 * N2; ++i) {
        A2[i] = static_cast<float>(rand()) / RAND_MAX;
    }
    for (int i = 0; i < N2 * P2; ++i) {
        B2[i] = static_cast<float>(rand()) / RAND_MAX;
    }
    matrixMultiply(A2, B2, C2, M2, N2, P2);
    verifyResult(A2, B2, C2, M2, N2, P2);
    free(A2);
    free(B2);
    free(C2);

    std::cout << "All tests passed successfully." << std::endl;

    return 0;
}
